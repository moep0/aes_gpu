#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <time.h>
#include <math.h>
#include <inttypes.h>
#include <iostream>
#include <fstream>
#include <cstring>
#include <iomanip>
#include <hip/hip_runtime.h>

//设置数据块长度16字节（128位）
//设置gpu中每block中thread数量512
#define AES_BLOCK_SIZE 16
#define THREADS_PER_BLOCK 512

// S盒，host，扩展密钥用
uint8_t s_box[256] = {
    0x63, 0x7c, 0x77, 0x7b, 0xf2, 0x6b, 0x6f, 0xc5,
    0x30, 0x01, 0x67, 0x2b, 0xfe, 0xd7, 0xab, 0x76,
    0xca, 0x82, 0xc9, 0x7d, 0xfa, 0x59, 0x47, 0xf0,
    0xad, 0xd4, 0xa2, 0xaf, 0x9c, 0xa4, 0x72, 0xc0,
    0xb7, 0xfd, 0x93, 0x26, 0x36, 0x3f, 0xf7, 0xcc,
    0x34, 0xa5, 0xe5, 0xf1, 0x71, 0xd8, 0x31, 0x15,
    0x04, 0xc7, 0x23, 0xc3, 0x18, 0x96, 0x05, 0x9a,
    0x07, 0x12, 0x80, 0xe2, 0xeb, 0x27, 0xb2, 0x75,
    0x09, 0x83, 0x2c, 0x1a, 0x1b, 0x6e, 0x5a, 0xa0,
    0x52, 0x3b, 0xd6, 0xb3, 0x29, 0xe3, 0x2f, 0x84,
    0x53, 0xd1, 0x00, 0xed, 0x20, 0xfc, 0xb1, 0x5b,
    0x6a, 0xcb, 0xbe, 0x39, 0x4a, 0x4c, 0x58, 0xcf,
    0xd0, 0xef, 0xaa, 0xfb, 0x43, 0x4d, 0x33, 0x85,
    0x45, 0xf9, 0x02, 0x7f, 0x50, 0x3c, 0x9f, 0xa8,
    0x51, 0xa3, 0x40, 0x8f, 0x92, 0x9d, 0x38, 0xf5,
    0xbc, 0xb6, 0xda, 0x21, 0x10, 0xff, 0xf3, 0xd2,
    0xcd, 0x0c, 0x13, 0xec, 0x5f, 0x97, 0x44, 0x17,
    0xc4, 0xa7, 0x7e, 0x3d, 0x64, 0x5d, 0x19, 0x73,
    0x60, 0x81, 0x4f, 0xdc, 0x22, 0x2a, 0x90, 0x88,
    0x46, 0xee, 0xb8, 0x14, 0xde, 0x5e, 0x0b, 0xdb,
    0xe0, 0x32, 0x3a, 0x0a, 0x49, 0x06, 0x24, 0x5c,
    0xc2, 0xd3, 0xac, 0x62, 0x91, 0x95, 0xe4, 0x79,
    0xe7, 0xc8, 0x37, 0x6d, 0x8d, 0xd5, 0x4e, 0xa9,
    0x6c, 0x56, 0xf4, 0xea, 0x65, 0x7a, 0xae, 0x08,
    0xba, 0x78, 0x25, 0x2e, 0x1c, 0xa6, 0xb4, 0xc6,
    0xe8, 0xdd, 0x74, 0x1f, 0x4b, 0xbd, 0x8b, 0x8a,
    0x70, 0x3e, 0xb5, 0x66, 0x48, 0x03, 0xf6, 0x0e,
    0x61, 0x35, 0x57, 0xb9, 0x86, 0xc1, 0x1d, 0x9e,
    0xe1, 0xf8, 0x98, 0x11, 0x69, 0xd9, 0x8e, 0x94,
    0x9b, 0x1e, 0x87, 0xe9, 0xce, 0x55, 0x28, 0xdf,
    0x8c, 0xa1, 0x89, 0x0d, 0xbf, 0xe6, 0x42, 0x68,
    0x41, 0x99, 0x2d, 0x0f, 0xb0, 0x54, 0xbb, 0x16
};

// S盒，device，加密用
__constant__ static const uint8_t sbox[256] = {
    0x63, 0x7c, 0x77, 0x7b, 0xf2, 0x6b, 0x6f, 0xc5,
    0x30, 0x01, 0x67, 0x2b, 0xfe, 0xd7, 0xab, 0x76,
    0xca, 0x82, 0xc9, 0x7d, 0xfa, 0x59, 0x47, 0xf0,
    0xad, 0xd4, 0xa2, 0xaf, 0x9c, 0xa4, 0x72, 0xc0,
    0xb7, 0xfd, 0x93, 0x26, 0x36, 0x3f, 0xf7, 0xcc,
    0x34, 0xa5, 0xe5, 0xf1, 0x71, 0xd8, 0x31, 0x15,
    0x04, 0xc7, 0x23, 0xc3, 0x18, 0x96, 0x05, 0x9a,
    0x07, 0x12, 0x80, 0xe2, 0xeb, 0x27, 0xb2, 0x75,
    0x09, 0x83, 0x2c, 0x1a, 0x1b, 0x6e, 0x5a, 0xa0,
    0x52, 0x3b, 0xd6, 0xb3, 0x29, 0xe3, 0x2f, 0x84,
    0x53, 0xd1, 0x00, 0xed, 0x20, 0xfc, 0xb1, 0x5b,
    0x6a, 0xcb, 0xbe, 0x39, 0x4a, 0x4c, 0x58, 0xcf,
    0xd0, 0xef, 0xaa, 0xfb, 0x43, 0x4d, 0x33, 0x85,
    0x45, 0xf9, 0x02, 0x7f, 0x50, 0x3c, 0x9f, 0xa8,
    0x51, 0xa3, 0x40, 0x8f, 0x92, 0x9d, 0x38, 0xf5,
    0xbc, 0xb6, 0xda, 0x21, 0x10, 0xff, 0xf3, 0xd2,
    0xcd, 0x0c, 0x13, 0xec, 0x5f, 0x97, 0x44, 0x17,
    0xc4, 0xa7, 0x7e, 0x3d, 0x64, 0x5d, 0x19, 0x73,
    0x60, 0x81, 0x4f, 0xdc, 0x22, 0x2a, 0x90, 0x88,
    0x46, 0xee, 0xb8, 0x14, 0xde, 0x5e, 0x0b, 0xdb,
    0xe0, 0x32, 0x3a, 0x0a, 0x49, 0x06, 0x24, 0x5c,
    0xc2, 0xd3, 0xac, 0x62, 0x91, 0x95, 0xe4, 0x79,
    0xe7, 0xc8, 0x37, 0x6d, 0x8d, 0xd5, 0x4e, 0xa9,
    0x6c, 0x56, 0xf4, 0xea, 0x65, 0x7a, 0xae, 0x08,
    0xba, 0x78, 0x25, 0x2e, 0x1c, 0xa6, 0xb4, 0xc6,
    0xe8, 0xdd, 0x74, 0x1f, 0x4b, 0xbd, 0x8b, 0x8a,
    0x70, 0x3e, 0xb5, 0x66, 0x48, 0x03, 0xf6, 0x0e,
    0x61, 0x35, 0x57, 0xb9, 0x86, 0xc1, 0x1d, 0x9e,
    0xe1, 0xf8, 0x98, 0x11, 0x69, 0xd9, 0x8e, 0x94,
    0x9b, 0x1e, 0x87, 0xe9, 0xce, 0x55, 0x28, 0xdf,
    0x8c, 0xa1, 0x89, 0x0d, 0xbf, 0xe6, 0x42, 0x68,
    0x41, 0x99, 0x2d, 0x0f, 0xb0, 0x54, 0xbb, 0x16
};


// 逆S盒，device，解密用
__constant__ static const uint8_t sboxinv[256] = {
    0x52, 0x09, 0x6a, 0xd5, 0x30, 0x36, 0xa5, 0x38,
    0xbf, 0x40, 0xa3, 0x9e, 0x81, 0xf3, 0xd7, 0xfb,
    0x7c, 0xe3, 0x39, 0x82, 0x9b, 0x2f, 0xff, 0x87,
    0x34, 0x8e, 0x43, 0x44, 0xc4, 0xde, 0xe9, 0xcb,
    0x54, 0x7b, 0x94, 0x32, 0xa6, 0xc2, 0x23, 0x3d,
    0xee, 0x4c, 0x95, 0x0b, 0x42, 0xfa, 0xc3, 0x4e,
    0x08, 0x2e, 0xa1, 0x66, 0x28, 0xd9, 0x24, 0xb2,
    0x76, 0x5b, 0xa2, 0x49, 0x6d, 0x8b, 0xd1, 0x25,
    0x72, 0xf8, 0xf6, 0x64, 0x86, 0x68, 0x98, 0x16,
    0xd4, 0xa4, 0x5c, 0xcc, 0x5d, 0x65, 0xb6, 0x92,
    0x6c, 0x70, 0x48, 0x50, 0xfd, 0xed, 0xb9, 0xda,
    0x5e, 0x15, 0x46, 0x57, 0xa7, 0x8d, 0x9d, 0x84,
    0x90, 0xd8, 0xab, 0x00, 0x8c, 0xbc, 0xd3, 0x0a,
    0xf7, 0xe4, 0x58, 0x05, 0xb8, 0xb3, 0x45, 0x06,
    0xd0, 0x2c, 0x1e, 0x8f, 0xca, 0x3f, 0x0f, 0x02,
    0xc1, 0xaf, 0xbd, 0x03, 0x01, 0x13, 0x8a, 0x6b,
    0x3a, 0x91, 0x11, 0x41, 0x4f, 0x67, 0xdc, 0xea,
    0x97, 0xf2, 0xcf, 0xce, 0xf0, 0xb4, 0xe6, 0x73,
    0x96, 0xac, 0x74, 0x22, 0xe7, 0xad, 0x35, 0x85,
    0xe2, 0xf9, 0x37, 0xe8, 0x1c, 0x75, 0xdf, 0x6e,
    0x47, 0xf1, 0x1a, 0x71, 0x1d, 0x29, 0xc5, 0x89,
    0x6f, 0xb7, 0x62, 0x0e, 0xaa, 0x18, 0xbe, 0x1b,
    0xfc, 0x56, 0x3e, 0x4b, 0xc6, 0xd2, 0x79, 0x20,
    0x9a, 0xdb, 0xc0, 0xfe, 0x78, 0xcd, 0x5a, 0xf4,
    0x1f, 0xdd, 0xa8, 0x33, 0x88, 0x07, 0xc7, 0x31,
    0xb1, 0x12, 0x10, 0x59, 0x27, 0x80, 0xec, 0x5f,
    0x60, 0x51, 0x7f, 0xa9, 0x19, 0xb5, 0x4a, 0x0d,
    0x2d, 0xe5, 0x7a, 0x9f, 0x93, 0xc9, 0x9c, 0xef,
    0xa0, 0xe0, 0x3b, 0x4d, 0xae, 0x2a, 0xf5, 0xb0,
    0xc8, 0xeb, 0xbb, 0x3c, 0x83, 0x53, 0x99, 0x61,
    0x17, 0x2b, 0x04, 0x7e, 0xba, 0x77, 0xd6, 0x26,
    0xe1, 0x69, 0x14, 0x63, 0x55, 0x21, 0x0c, 0x7d
};


// x-time operation
__device__ uint8_t rj_xtime(uint8_t x){
  return (x & 0x80) ? ((x << 1) ^ 0x1b) : (x << 1);
}


// 字节代换
__device__ void aes_subBytes(uint8_t *buf){
  register uint8_t i, b;
  for (i = 0; i < 16; ++i){
    b = buf[i];
    buf[i] = sbox[b];
  }
} 


// 逆字节代换
__device__ void aes_subBytes_inv(uint8_t *buf){
  register uint8_t i, b;
  for (i = 0; i < 16; ++i){
    b = buf[i];
    buf[i] = sboxinv[b];
  }
} 


// 轮密钥加
__device__ void aes_addRoundKey(uint8_t *buf, uint8_t *key,uint8_t r){
  //register uint8_t i = 16;
  //while (i--){
   buf[16] ^= key[16*r+16];
   buf[15] ^= key[16*r+15];
   buf[14] ^= key[16*r+14];
   buf[13] ^= key[16*r+13];
   buf[12] ^= key[16*r+12];
   buf[11] ^= key[16*r+11];
   buf[10] ^= key[16*r+10];
   buf[9] ^= key[16*r+9];
   buf[8] ^= key[16*r+8];
   buf[7] ^= key[16*r+7];
   buf[6] ^= key[16*r+6];
   buf[5] ^= key[16*r+5];
   buf[4] ^= key[16*r+4];
   buf[3] ^= key[16*r+3];
   buf[2] ^= key[16*r+2];
   buf[1] ^= key[16*r+1];
   buf[0] ^= key[16*r+0];
  //}
} 

//行位移
__device__ void aes_shiftRows(uint8_t *buf){
  register uint8_t i, j; 
  i = buf[1];
  buf[1] = buf[5];
  buf[5] = buf[9];
  buf[9] = buf[13];
  buf[13] = i;
  i = buf[10];
  buf[10] = buf[2];
  buf[2] = i;
  j = buf[3];
  buf[3] = buf[15];
  buf[15] = buf[11];
  buf[11] = buf[7];
  buf[7] = j;
  j = buf[14];
  buf[14] = buf[6];
  buf[6]  = j;
}


// 逆行位移
__device__ void aes_shiftRows_inv(uint8_t *buf){
  register uint8_t i, j; 
  i = buf[1];
  buf[1] = buf[13];
  buf[13] = buf[9];
  buf[9] = buf[5];
  buf[5] = i;
  i = buf[2];
  buf[2] = buf[10];
  buf[10] = i;
  j = buf[3];
  buf[3] = buf[7];
  buf[7] = buf[11];
  buf[11] = buf[15];
  buf[15] = j;
  j = buf[6];
  buf[6] = buf[14];
  buf[14] = j;
} 

// 列混合
__device__ void aes_mixColumns(uint8_t *buf){
  register uint8_t i, a, b, c, d, e;
  for (i = 0; i < 16; i += 4){
    a = buf[i];
    b = buf[i + 1];
    c = buf[i + 2];
    d = buf[i + 3];
    e = a ^ b ^ c ^ d;
    buf[i] ^= e ^ rj_xtime(a^b);
    buf[i+1] ^= e ^ rj_xtime(b^c);
    buf[i+2] ^= e ^ rj_xtime(c^d);
    buf[i+3] ^= e ^ rj_xtime(d^a);
  }
} 


// 逆列混合
__device__ void aes_mixColumns_inv(uint8_t *buf){
  register uint8_t i, a, b, c, d, e, x, y, z;
  for (i = 0; i < 16; i += 4){
    a = buf[i];
    b = buf[i + 1];
    c = buf[i + 2];
    d = buf[i + 3];
    e = a ^ b ^ c ^ d;
    z = rj_xtime(e);
    x = e ^ rj_xtime(rj_xtime(z^a^c));
    y = e ^ rj_xtime(rj_xtime(z^b^d));
    buf[i] ^= x ^ rj_xtime(a^b);
    buf[i+1] ^= y ^ rj_xtime(b^c);
    buf[i+2] ^= x ^ rj_xtime(c^d);
    buf[i+3] ^= y ^ rj_xtime(d^a);
  }
} 

// 字循环 用于密钥扩展
void rot_word(uint8_t *w) {
  uint8_t tmp;
  uint8_t i;
  tmp = w[0];
  for (i = 0; i < 3; i++) {
    w[i] = w[i+1];
  }
  w[3] = tmp;
}

// 字节代换 用于密钥扩展
void sub_word(uint8_t *w) {
  uint8_t i;
  for (i = 0; i < 4; i++) {
    w[i] = s_box[w[i]];
  }
}

// 密钥扩展
void aes_key_expansion(uint8_t *key, uint8_t *w) {
  uint8_t tmp[4];
  uint8_t i;
  int Nb=4;
  int Nr=14;
  int Nk=8;
  uint8_t len = Nb*(Nr+1);
  uint8_t RC[11] = {0x00,0x01,0x02,0x04,0x08,0x10,0x20,0x40,0x80,0x1B,0x36};

  for (i = 0; i < Nk; i++) {
    w[4*i+0] = key[4*i+0];
    w[4*i+1] = key[4*i+1];
    w[4*i+2] = key[4*i+2];
    w[4*i+3] = key[4*i+3];
  }

  for (i = Nk; i < len; i++) {
    tmp[0] = w[4*(i-1)+0];
    tmp[1] = w[4*(i-1)+1];
    tmp[2] = w[4*(i-1)+2];
    tmp[3] = w[4*(i-1)+3];

    if (i%Nk == 0) {
      rot_word(tmp);
      sub_word(tmp);
      tmp[0] = RC[i/Nk]^tmp[0];
      tmp[1] = 0x00^tmp[1];
      tmp[2] = 0x00^tmp[2];
      tmp[3] = 0x00^tmp[3];
    } else if (Nk > 6 && i%Nk == 4) {
      sub_word(tmp);
    }

    w[4*i+0] = w[4*(i-Nk)+0]^tmp[0];
    w[4*i+1] = w[4*(i-Nk)+1]^tmp[1];
    w[4*i+2] = w[4*(i-Nk)+2]^tmp[2];
    w[4*i+3] = w[4*(i-Nk)+3]^tmp[3];
  }

}


// 使用aes加密算法，对一块（128位）数据进行加密 
__constant__ uint8_t w2[240];
__constant__ uint8_t w3[240];
__global__ void aes256_encrypt_ecb(uint8_t *buf_d, unsigned long numbytes){
  uint8_t *key=w2;
  uint8_t i;
  uint8_t buf_t[AES_BLOCK_SIZE]; // thread buffer
  //计算待加密数据在总数据中的偏移
  unsigned long offset = (blockIdx.x * THREADS_PER_BLOCK * AES_BLOCK_SIZE) + (threadIdx.x * AES_BLOCK_SIZE);
  if (offset >= numbytes) {  return; }
  //拷贝待加密数据至buf_t
  memcpy(buf_t, &buf_d[offset], AES_BLOCK_SIZE);

  //加密，共14轮
  aes_addRoundKey(buf_t, key,0);
  for(i = 1; i < 14; i++){
    aes_subBytes(buf_t);
    aes_shiftRows(buf_t);
    aes_mixColumns(buf_t);
    aes_addRoundKey( buf_t, key,i);
  }
  aes_subBytes(buf_t);
  aes_shiftRows(buf_t);
  aes_addRoundKey(buf_t,key,14);
  //将加密后的buf_t拷贝回总数据
  memcpy(&buf_d[offset], buf_t, AES_BLOCK_SIZE);
  __syncthreads();
} 



// 使用aes解密算法，对一块（128位）数据进行解密
__global__ void aes256_decrypt_ecb(uint8_t *buf_d, unsigned long numbytes){
  uint8_t i;
  uint8_t *key=w3;
  uint8_t buf_t[AES_BLOCK_SIZE];
  //计算待解密数据在总数据中的偏移
  unsigned long offset = (blockIdx.x * THREADS_PER_BLOCK * AES_BLOCK_SIZE) + (threadIdx.x * AES_BLOCK_SIZE);
  if (offset >= numbytes) { return; }
  //拷贝待解密数据至buf_t
  memcpy(buf_t, &buf_d[offset], AES_BLOCK_SIZE);

  //解密，共14轮
  aes_addRoundKey(buf_t, key,14);
  for (i = 1; i < 14; i++){
  	aes_shiftRows_inv(buf_t);
  	aes_subBytes_inv(buf_t);
  	aes_addRoundKey( buf_t,  key,14-i);
    aes_mixColumns_inv(buf_t);
    }
  aes_shiftRows_inv(buf_t);
  aes_subBytes_inv(buf_t);
  aes_addRoundKey( buf_t,  key,0);
  //将解密后的buf_t拷贝回总数据
  memcpy(&buf_d[offset], buf_t, AES_BLOCK_SIZE);
  __syncthreads();
} 



//aes加密
void encryptdemo(uint8_t *key, uint8_t *buf, unsigned long numbytes){
  uint8_t *buf_d;

  uint8_t *w;
  const int nStreams = 16;
  const int ChunkSize = numbytes / nStreams;
  
  printf("\nBeginning encryption\n");

  //记录加密算法开始时间
  hipEvent_t start1;
  hipEventCreate(&start1);
  hipEvent_t stop1;
  hipEventCreate(&stop1);
  hipEventRecord(start1);
  //创建工作流 
  hipStream_t streams[nStreams];
  for (int i = 0; i < nStreams; i++) {
    hipStreamCreate(&streams[i]);
  }
  //将s盒拷贝到常量内存中，可以加快读取速度。
  hipMemcpyToSymbol(HIP_SYMBOL(sbox), sbox, sizeof(uint8_t)*256);
  w = (uint8_t*)malloc(240*sizeof(uint8_t));
  //密钥扩展
  aes_key_expansion(key, w);
  hipMalloc((void**)&buf_d, numbytes);
  //将扩展后的密钥存储到常量内存中
  hipMemcpyToSymbol(HIP_SYMBOL(w2), w, 240*sizeof(uint8_t));
  //计算dimblock数量，需要多除一个工作流的总数
  dim3 dimBlock(ceil((double)numbytes/nStreams / (double)(THREADS_PER_BLOCK * AES_BLOCK_SIZE)));
  dim3 dimGrid(THREADS_PER_BLOCK);
  printf("Creating %d threads over %d blocks\n", dimBlock.x*dimGrid.x, dimBlock.x);

  int nOffset = 0;
  //开始工作流
  for(int i=0; i<nStreams; i++)
  {
    
    nOffset = ChunkSize*i;

    hipMemcpyAsync(  buf_d+nOffset,
                      buf+nOffset,
                      ChunkSize,
                      hipMemcpyHostToDevice,
                      streams[i] );


    aes256_encrypt_ecb<<<dimBlock, dimGrid, 0, streams[i]>>>(
                                   buf_d+nOffset, 
                                   ChunkSize);

    hipMemcpyAsync(  buf+nOffset,
                      buf_d+nOffset,
                      ChunkSize,
                      hipMemcpyDeviceToHost,
                      streams[i] );
      
  }
  //等待运算完成
  hipDeviceSynchronize();

  //记录加密算法结束时间，并计算加密速度  
  hipEventRecord(stop1);
  hipEventSynchronize(stop1);
  float msecTotal1,total;
  hipEventElapsedTime(&msecTotal1, start1, stop1);
  total=msecTotal1/1000;
  printf("time:%f\n",total);
  printf("Throughput: %f Gbps\n", numbytes/total/1024/1024/1024*8);

}


// aes解密
void decryptdemo(uint8_t *key, uint8_t *buf, unsigned long numbytes){
  uint8_t *buf_d;

  uint8_t *w;
  const int nStreams = 16;
  const int ChunkSize = numbytes / nStreams;
  
  printf("\nBeginning encryption\n");

  //记录解密算法开始时间
  hipEvent_t start1;
  hipEventCreate(&start1);
  hipEvent_t stop1;
  hipEventCreate(&stop1);
  hipEventRecord(start1);
  //创建工作流  
  hipStream_t streams[nStreams];
  for (int i = 0; i < nStreams; i++) {
    hipStreamCreate(&streams[i]);
  }
  //将s盒拷贝到常量内存中，可以加快读取速度。
  hipMemcpyToSymbol(HIP_SYMBOL(sbox), sbox, sizeof(uint8_t)*256);
  w = (uint8_t*)malloc(240*sizeof(uint8_t));
  //密钥扩展
  aes_key_expansion(key, w);
  hipMalloc((void**)&buf_d, numbytes);
  //将扩展后的密钥存储到常量内存中
  hipMemcpyToSymbol(HIP_SYMBOL(w3), w, 240*sizeof(uint8_t));
  //计算dimblock数量，需要多除一个工作流的总数
  dim3 dimBlock(ceil((double)numbytes/nStreams / (double)(THREADS_PER_BLOCK * AES_BLOCK_SIZE)));
  dim3 dimGrid(THREADS_PER_BLOCK);
  printf("Creating %d threads over %d blocks\n", dimBlock.x*dimGrid.x, dimBlock.x);

  int nOffset = 0;
  //开始工作流
  for(int i=0; i<nStreams; i++)
  {
    
    nOffset = ChunkSize*i;

    hipMemcpyAsync(  buf_d+nOffset,
                      buf+nOffset,
                      ChunkSize,
                      hipMemcpyHostToDevice,
                      streams[i] );


    aes256_decrypt_ecb<<<dimBlock, dimGrid, 0, streams[i]>>>(
                                   buf_d+nOffset, 
                                   ChunkSize);

    hipMemcpyAsync(  buf+nOffset,
                      buf_d+nOffset,
                      ChunkSize,
                      hipMemcpyDeviceToHost,
                      streams[i] );
      
  }
  //等待运算完成
  hipDeviceSynchronize();

  //记录解密算法结束时间，并计算解密速度
  hipEventRecord(stop1);
  hipEventSynchronize(stop1);
  float msecTotal1,total;
  hipEventElapsedTime(&msecTotal1, start1, stop1);
  total=msecTotal1/1000;
  printf("time:%f\n",total);
  printf("Throughput: %f Gbps\n", numbytes/total/1024/1024/1024*8);

}

__global__ void GPU_init() { }



int main(int argc,char** argv){

  FILE *file;
  uint8_t *buf,*buf2; 
  unsigned long numbytes;
  char *fname;
  int  i;
  int padding;
 
  uint8_t key[32];
  // 设置gpu
  int deviceCount = 0;
  hipError_t error_id = hipGetDeviceCount(&deviceCount);
  if (error_id != hipSuccess){
    printf("Error: %s\n", hipGetErrorString(error_id));
    printf("Exiting...\n");
    exit(EXIT_FAILURE);
  }
  if (deviceCount == 0){
    printf("There are no available device(s) that support CUDA\n");
    exit(EXIT_FAILURE);
  }


  // 打开待加密文件
  fname = argv[1];  
  file = fopen(fname, "r");
  if (file == NULL) {printf("File %s doesn't exist\n", fname); exit(1); }
  printf("Opened file %s\n", fname);
  fseek(file, 0L, SEEK_END);
  numbytes = ftell(file);
  printf("Size is %lu\n", numbytes);

  // 将待加密数据读取到内存
  fseek(file, 0L, SEEK_SET);
  buf = (uint8_t*)calloc(numbytes, sizeof(uint8_t)); 
  if(buf == NULL) exit(1);
  if (fread(buf, 1, numbytes, file) != numbytes)
  {
    printf("Unable to read all bytes from file %s\n", fname);
    exit(EXIT_FAILURE);
  }
  fclose(file);

  // 补全
  padding = AES_BLOCK_SIZE * 16 - numbytes % (AES_BLOCK_SIZE * 16);
  numbytes += padding;
  printf("Padding file with %d bytes for a new size of %lu\n", padding, numbytes);

  // 生成密钥
  for (i = 0; i < sizeof(key);i++) key[i] = i;
  hipHostMalloc((void**)&buf2, numbytes, hipHostMallocDefault);
  hipMemcpy(buf2, buf, numbytes, hipMemcpyHostToHost);
  // gpu初始化
  GPU_init<<<1, 1>>>();


  // 调用加密算法
  encryptdemo(key, buf2, numbytes);
  // 将加密后的数据写入cipher.txt
  file = fopen("cipher.txt", "w");
  fwrite(buf2, 1, numbytes, file);
  fclose(file);

  // 解密
  decryptdemo(key, buf2, numbytes);
  // 将解密后的数据写回output.txt
  file = fopen("output.txt", "w");
  fwrite(buf2, 1, numbytes - padding, file);
  fclose(file);
  free(buf);
  //free(buf2);

  return EXIT_SUCCESS;
}